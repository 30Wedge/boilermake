#include "hip/hip_runtime.h"
#include "mouse.hh"
#include <stdlib.h>
#include <iostream>

Mouse::Mouse (std::string name)
    :
    Animal(name)
{
    m_sound = "Squeak!";
}

void randomBlock(int* vec, size_t s)
{
    for(size_t i = 0; i < s; i++)
        vec[i] = rand() % 5; 
}

__global__ void add_vec(int* a, int* b, int* c, size_t size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    while( tid < size)
    {
        c[tid] = a[tid] + b[tid];
        tid += gridDim.x * blockDim.x;
    }    
}

//capsule to run cuda Kernel 
void runKernel()
{
    size_t s = 100;
    int* a_dev, *b_dev, *c_dev;

    int* a = (int*) malloc(sizeof(int) * s);
    randomBlock(a, s);
    int* b = (int*) malloc(sizeof(int) * s);
    randomBlock(b, s);
    int* c = (int*) malloc(sizeof(int) * s);
    randomBlock(c, s);

    hipMalloc((void **) &a_dev, sizeof(int) * s);
    hipMemcpy(a_dev, a, sizeof(int) * s, hipMemcpyHostToDevice);
    hipMalloc((void **) &b_dev, sizeof(int) * s);
    hipMemcpy(b_dev, b, sizeof(int) * s, hipMemcpyHostToDevice);
    hipMalloc((void **) &c_dev, sizeof(int) * s);

    add_vec<<<1, 100>>>(a_dev, b_dev, c_dev, s);

    hipMemcpy(c, c_dev, sizeof(int) * s, hipMemcpyDeviceToHost);

    std::cout << "My mouse brain does math too: ";
    for(int i = 0; i < 10; i++)
        std::cout << a[i] << " + " << b[i] << " = " << c[i] << "; ";
    std::cout << std::endl;
}

//mouse is also cuda-powered 
void Mouse::talk()
{
    Animal::talk(); //keep up the squeaks
    
    runKernel();
}

